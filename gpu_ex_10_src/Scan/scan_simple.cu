#include "hip/hip_runtime.h"

#include <stdio.h>
#include "hip/hip_runtime.h"
#include <GL/glut.h>

#define N 512

GLfloat viewPosition[4] = { 0.0f, 5.0f, 10.0f, 1.0f };
GLfloat viewDirection[4] = { -0.0f, -5.0f, -10.0f, 0.0f };
GLfloat viewAngle = 45.0f;
GLfloat viewNear = 4.5f;
GLfloat viewFar = 25.0f;

GLfloat xRotationAngle = 0.0f;
GLfloat yRotationAngle = 0.0f;

GLfloat xRotationSpeed = 3.0f;
GLfloat yRotationSpeed = 4.5f;

GLfloat depthPixels[N*N];
GLfloat colorPixels[N*N];
GLfloat filteredPixels[N*N];

float focusDepth = 0.5f;
float sizeScale = 20.0f;

float *devColorPixelsSrc, *devColorPixelsDst, *devDepthPixels;

void drawGround()
{
	GLfloat grey[3] = { 0.8f, 0.8f, 0.8f };

	glNormal3f(0, 1, 0);
	glMaterialfv(GL_FRONT, GL_AMBIENT, grey);
	glMaterialfv(GL_FRONT, GL_DIFFUSE, grey);
	glBegin(GL_QUADS);
	glVertex3f(-10, 0, 10);
	glVertex3f(10, 0, 10);
	glVertex3f(10, 0, -10);
	glVertex3f(-10, 0, -10);
	glEnd();
}


void drawScene()
{
	GLfloat diffuse1[4] = { 0.5f, 0.5f, 0.5f, 1.0f };
	GLfloat lightAmbient[4] = { 0.0f, 0.0f, 0.0f, 1.0f };
	GLfloat lightDiffuse[4] = { 0.2f, 0.2f, 0.2f, 1.0f };
	GLfloat lightPosition[4] = { 0.5f, 10.5f, 6.0f, 1.0f };

	glLightfv(GL_LIGHT0, GL_AMBIENT, lightAmbient);
	glLightfv(GL_LIGHT0, GL_POSITION, lightPosition);
	glLightfv(GL_LIGHT0, GL_DIFFUSE, lightDiffuse);

	glMaterialfv(GL_FRONT_AND_BACK, GL_AMBIENT, diffuse1);
	glMaterialfv(GL_FRONT_AND_BACK, GL_DIFFUSE, diffuse1);
	glPushMatrix();
	glTranslatef(0.0f, 1.0f, 0.0f);
	glRotatef(-yRotationAngle / 3.0f, 0.0f, 1.0f, 0.0f);
	glutSolidTeapot(1.0f);
	glPopMatrix();

	glMaterialfv(GL_FRONT_AND_BACK, GL_AMBIENT, diffuse1);
	glMaterialfv(GL_FRONT_AND_BACK, GL_DIFFUSE, diffuse1);
	glPushMatrix();
	glTranslatef(-1.0f, 1.0f, 3.0f);
	glRotatef(-yRotationAngle / 3.0f, 0.0f, 1.0f, 0.0f);
	glutSolidTeapot(1.0f);
	glPopMatrix();

	glMaterialfv(GL_FRONT_AND_BACK, GL_AMBIENT, diffuse1);
	glMaterialfv(GL_FRONT_AND_BACK, GL_DIFFUSE, diffuse1);
	glPushMatrix();
	glTranslatef(1.0f, 1.0f, -3.0f);
	glRotatef(-yRotationAngle / 3.0f, 0.0f, 1.0f, 0.0f);
	glutSolidTeapot(1.0f);
	glPopMatrix();

	glMaterialfv(GL_FRONT_AND_BACK, GL_AMBIENT, diffuse1);
	glMaterialfv(GL_FRONT_AND_BACK, GL_DIFFUSE, diffuse1);
	glPushMatrix();
	glTranslatef(-2.0f, 1.0f, 6.0f);
	glRotatef(-yRotationAngle / 3.0f, 0.0f, 1.0f, 0.0f);
	glutSolidTeapot(1.0f);
	glPopMatrix();

	glMaterialfv(GL_FRONT_AND_BACK, GL_AMBIENT, diffuse1);
	glMaterialfv(GL_FRONT_AND_BACK, GL_DIFFUSE, diffuse1);
	glPushMatrix();
	glTranslatef(2.0f, 1.0f, -6.0f);
	glRotatef(-yRotationAngle / 3.0f, 0.0f, 1.0f, 0.0f);
	glutSolidTeapot(1.0f);
	glPopMatrix();

	drawGround();

}



void initGL()
{
	glEnable(GL_DEPTH_TEST);
	glEnable(GL_LIGHTING);               // OpenGL Lichtquellen aktivieren
	glEnable(GL_LIGHT0);                 // Lichtquelle 0 anmachen 

	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	gluPerspective(viewAngle, 1.0f, viewNear, viewFar);
	glMatrixMode(GL_MODELVIEW);
	glLoadIdentity();
}


// DONE: Transpose kernel
__global__ void transpose(float *dstImage, float *srcImage)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int index = y + x * gridDim.x;

	int targetIndex = x + y *gridDim.x;

	dstImage[index] = srcImage[targetIndex];
}




__global__ void sat_filter(float *dstImage, float *sat, float *srcDepth,
	float focusDepth, float sizeScale, int n)
{
	int x = blockIdx.x;
	int y = threadIdx.x;
	int index = x + y * n;

	// DONE: Filtergr��e bestimmen
	int filterSize = 1 + sizeScale * abs(srcDepth[index] - focusDepth);

	// DONE: Anzahl der Pixel im Filterkern bestimmen	

	int min_x = min(0, x - filterSize);
	int max_x = max(0, x + filterSize - n + 1);

	int min_y = min(0, y - filterSize);
	int max_y = max(0, y + filterSize - n + 1);

	int numpixel = (2 * filterSize + min_x - max_x) * (2 * filterSize - max_y + min_y);

	// DONE: SAT-Werte f�r die Eckpunkte des Filterkerns bestimmen.


	float A = sat[index + (filterSize - max_x) + (filterSize - max_y) * n];	// top right



	float B = sat[index - (filterSize + min_x) + (filterSize - max_y) * n];

	// bottom right

	float C = sat[index + (filterSize - max_x) - (filterSize + min_y) * n];
	// bottom lef

	float D = sat[index - (filterSize + min_x) - (filterSize + min_y) * n];

	// DONE: Mittelwert berechnen.
	dstImage[index] = (A - B - C + D) / numpixel;
}


/*
__global__ void sat_filter(float *dstImage, float *sat, float *srcDepth,
float focusDepth, float sizeScale, int n)
{
int x = threadIdx.x + blockIdx.x * blockDim.x;
int y = threadIdx.y + blockIdx.y * blockDim.y;
int index = y + x * gridDim.x;

// DONE: Filtergr��e bestimmen
int filterSize = int(1.f + sizeScale * fabsf(srcDepth[index] - focusDepth));

// DONE: Anzahl der Pixel im Filterkern bestimmen
//see presentation 10 slide 42: w*h
float sumFilter = float(filterSize* filterSize);

// DONE: SAT-Werte f�r die Eckpunkte des Filterkerns bestimmen.
float A = sat[index];	// top right
float B = 0.f;
float C = 0.f;
float D = 0.f;

// borders:
// top left
if (x - filterSize >= 0)
B = sat[index - filterSize];
// bottom right
if (y + filterSize < N)
C = sat[index + filterSize * blockDim.x];
// bottom left
if (x - filterSize >= 0 && y + filterSize < N)
D = sat[index - filterSize + filterSize * blockDim.x];

// DONE: Mittelwert berechnen.
dstImage[index] = (A - B - C + D) / sumFilter;
}
*/



__global__ void scan_naive(float *g_odata, float *g_idata, int n)
{
	// Dynamically allocated shared memory for scan kernels
	__shared__  float temp[2 * N];

	int thid = threadIdx.x;
	int bid = blockIdx.x;

	int pout = 0;
	int pin = 1;

	// Cache the computational window in shared memory
	temp[pout*n + thid] = (thid > 0) ? g_idata[bid * N + thid - 1] : 0;

	for (int offset = 1; offset < n; offset *= 2)
	{
		pout = 1 - pout;
		pin = 1 - pout;
		__syncthreads();

		temp[pout*n + thid] = temp[pin*n + thid];

		if (thid >= offset)
			temp[pout*n + thid] += temp[pin*n + thid - offset];
	}

	__syncthreads();

	g_odata[bid * N + thid] = temp[pout*n + thid];
}


void initCUDA()
{
	hipMalloc((void**)&devColorPixelsSrc, N * N * sizeof(float));
	hipMalloc((void**)&devColorPixelsDst, N * N * sizeof(float));
	hipMalloc((void**)&devDepthPixels, N * N * sizeof(float));
}

void special(int key, int x, int y)
{
	switch (key) {
	case GLUT_KEY_UP:
		focusDepth += 0.05f;
		if (focusDepth > 1.0f) focusDepth = 1.0;
		break;
	case GLUT_KEY_DOWN:
		focusDepth -= 0.05f;
		if (focusDepth < 0.0f) focusDepth = 0.0;
		break;
	case GLUT_KEY_LEFT:
		sizeScale -= 1.0f;
		if (sizeScale > 100.0f) sizeScale = 100.0;
		break;
	case GLUT_KEY_RIGHT:
		sizeScale += 1.0f;
		if (sizeScale < 1.0f) sizeScale = 1.0;
		break;
	case GLUT_KEY_PAGE_UP:
		viewFar += 1.0f;
		initGL();
		break;
	case GLUT_KEY_PAGE_DOWN:
		viewFar -= 1.0f;
		if (viewFar < viewNear) viewFar = viewNear;
		initGL();
		break;
	}
}

void display(void)
{
	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

	// Szene rendern
	glLoadIdentity();
	gluLookAt(viewPosition[0], viewPosition[1], viewPosition[2],
		viewDirection[0] - viewPosition[0], viewDirection[1] - viewPosition[1], viewDirection[2] - viewPosition[2],
		0, 1, 0);
	drawScene();

	// Tiefe und Farbe in den RAM streamen.
	glReadPixels(0, 0, N, N, GL_DEPTH_COMPONENT, GL_FLOAT, depthPixels);
	glReadPixels(0, 0, N, N, GL_LUMINANCE, GL_FLOAT, colorPixels);

	// Beide arrays in den Device-Memory kopieren.
	hipMemcpy(devColorPixelsSrc, colorPixels, N * N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(devDepthPixels, depthPixels, N * N * sizeof(float), hipMemcpyHostToDevice);

	dim3 a(N, N, 1);

	// DONE: Scan    
	scan_naive << <N, N >> > (devColorPixelsDst, devColorPixelsSrc, N);


	// DONE: Transponieren 
	transpose << <a, 1 >> >(devColorPixelsSrc, devColorPixelsDst); //transposes image only if, #if is set to 1, no idea why, but it works! -- of course, otherwise the filter is not activated... m�hhhhhhh :P

	// DONE: Scan  
	scan_naive << <N, N >> > (devColorPixelsDst, devColorPixelsSrc, N); //read from transposed matrix, overwrite old source


	//DONE : Transponieren
	transpose << <a, 1 >> >(devColorPixelsSrc, devColorPixelsDst);

	// DONE: SAT-Filter anwenden
	sat_filter << <N, N >> > (devColorPixelsDst, devColorPixelsSrc, devDepthPixels, focusDepth, sizeScale, N);

	// Ergebnis in Host-Memory kopieren
	hipMemcpy(filteredPixels, devColorPixelsDst, N*N * 4, hipMemcpyDeviceToHost); // edited to write copy the correct correct output pointer

	// DONE: Beim #if aus der 0 eine 1 machen, damit das gefilterte Bild angezeigt wird!
#if 1
	// Mittelwert-Bild rendern
	glDrawPixels(N, N, GL_LUMINANCE, GL_FLOAT, filteredPixels);
#else
	// Durchreichen des Eingabebildes.
	glDrawPixels(N, N, GL_LUMINANCE, GL_FLOAT, colorPixels);
#endif

	xRotationAngle += xRotationSpeed;   // Rotationswinkel erhoehen
	yRotationAngle += yRotationSpeed;

	glutSwapBuffers();
}

int main(int argc, char **argv)
{
	glutInit(&argc, argv);
	glutInitDisplayMode(GLUT_RGB | GLUT_DEPTH | GLUT_DOUBLE);
	glutInitWindowSize(N, N);
	glutCreateWindow("Simple CUDA SAT Depth of Field");

	glutDisplayFunc(display);
	glutIdleFunc(display);
	glutSpecialFunc(special);

	initGL();
	initCUDA();

	glutMainLoop();

	return 0;
}