#include "hip/hip_runtime.h"

// Includes
#include <stdio.h>

// Type of the array in which we search for the maximum.
// If you use float, don't forget to type %f in the printf later on..
#define TYPE int

//#define USE_NAIVE

// Variables
TYPE* h_A;
TYPE* d_A;

// Functions
void Cleanup(void);
void WorstCaseInit(TYPE*, int);

__device__ __host__ TYPE cumax(TYPE a, TYPE b)
{
	return a > b ? a : b;
}

// Schema des naiven Ansatz
// o o o o o o o o  n=1
// |/  |/  |/  |/
// o   o   o   o    n=2
// |  /    |  /
// | /     | /
// o       o        n=4
// |      /
// |    /
// |  /
// |/
// o				Ergebnis

__global__ void reduce_max_naive(TYPE* A, int n)
{
	int i = blockIdx.x * n;
	A[2*i] = cumax( A[2*i], A[2*i+n]);
}

__global__ void reduce_max_not_naive(TYPE* A)
{
	//index, extern to specify size at runtime
	extern __shared__ TYPE cache[];

	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*(blockDim.x * 2) + threadIdx.x;
	cache[tid] =  cumax(A[i], A[i + blockDim.x]);
	__syncthreads();

	for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
	{	
		if (tid < s)
		{
			cache[tid] = cumax(cache[tid], cache[tid + s]);
		}
		__syncthreads();
		
	}

	/*if (tid < 32)
	{
		cache[tid] = cumax(cache[tid], cache[tid + 32]);
		cache[tid] = cumax(cache[tid], cache[tid + 16]);
		cache[tid] = cumax(cache[tid], cache[tid + 8]);
		cache[tid] = cumax(cache[tid], cache[tid + 4]);
		cache[tid] = cumax(cache[tid], cache[tid + 2]);
		cache[tid] = cumax(cache[tid], cache[tid + 1]);
	}*/

	if (tid == 0)
	{
		A[blockIdx.x] = cache[0];
	}
}

// Host code
int main(int argc, char** argv)
{
	printf("Reduce\n");
	int N = 1 << 15;
	int Nh = N / 2;
	size_t size = N * sizeof(TYPE);

	// Allocate input vector h_A
	h_A = (TYPE*)malloc(size);

	// Initialize input vector
	WorstCaseInit(h_A, N);

	// Allocate vector in device memory
	hipMalloc((void**)&d_A, size);

	// Copy vector from host memory to device memory
	hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);

	int maxThreadsPerBlock = prop.maxThreadsPerBlock;
	int threads = maxThreadsPerBlock /4;
	// Start tracking of elapsed time.
	hipEvent_t     start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

#ifdef USE_NAIVE	// Naive approach

	for (int n=1; n<N; n*=2)
		reduce_max_naive<<<Nh/n,1>>>(d_A, n);	

#else				// Better approach
	for (int i = 1; i < N; i *=threads)
	{
		reduce_max_not_naive << < N / threads/2, threads, sizeof(TYPE) * threads>> >(d_A);

	}
#endif

	// End tracking of elapsed time.
	hipEventRecord( stop, 0 );
	hipEventSynchronize( stop );
	float   elapsedTime;
	hipEventElapsedTime( &elapsedTime, start, stop );
	printf( "Time: %f ms\n", elapsedTime );
	hipEventDestroy( start );
	hipEventDestroy( stop );

	// Find the truth... :)
	TYPE maximum = 0;
	for (int i = 0; i < N; ++i) 	
		maximum = cumax(h_A[i], maximum);
	
	// Copy result (first element only) from device memory to host memory
    hipMemcpy(h_A, d_A, sizeof(TYPE), hipMemcpyDeviceToHost);

	// Validate result from GPU.
	if (maximum == h_A[0])
		printf("PASSED: %i == %i", maximum, h_A[0]);
	else printf("FAILED: %i != %i", maximum, h_A[0]);
    
    Cleanup();
}

void Cleanup(void)
{
    // Free device memory
    if (d_A)
        hipFree(d_A);
  
    // Free host memory
    if (h_A)
        free(h_A);  
        
    hipDeviceReset();
        
    printf("\nPress ENTER to exit...\n");
    fflush( stdout);
    fflush( stderr);
    getchar();    

    exit(0);
}

void WorstCaseInit(TYPE* data, int n)
{
	// Using a list sorted in ascending order is the worst case.
    for (int i = 0; i < n; ++i)
		data[i] = (TYPE)(i);
}
