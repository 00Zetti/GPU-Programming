#include "hip/hip_runtime.h"

// Includes
#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "common.h"

// Variables
float* h_A;
float* h_B;
float* h_C;
float* d_A;
float* d_B;
float* d_C;

// DONE: Add Variables
float* h_D;
float* h_E;
float* d_D;
float* d_E;

// Functions
void Cleanup(void);
void RandomInit(float*, int);

// Device code
__global__ void VecAdd(const float* A, const float* B, float* C)
{
    int i = threadIdx.x;    
    C[i] = A[i] + B[i];	
}

// DONE: Kernel 2
__global__ void VecAdd2(const float* A, const float* B, float* C, const float* D, float* E)
{
	int i = threadIdx.x;
	C[i] = A[i] + B[i];
	E[i] = A[i] + B[i] * D[i];
}

// Host code
int main(int argc, char** argv)
{
    printf("Simple vector addition\n");
    int N = 256;
    size_t size = N * sizeof(float);    

    // Allocate input vectors h_A, h_B and h_C in host memory
    h_A = (float*)malloc(size);
    if (h_A == 0) Cleanup();
    h_B = (float*)malloc(size);
    if (h_B == 0) Cleanup();
    h_C = (float*)malloc(size);
    if (h_C == 0) Cleanup();

	// DONE: Allocate host memory for h_D and h_E
	h_D = (float*)malloc(size);
	if (h_D == 0) Cleanup();
	h_E = (float*)malloc(size);
	if (h_E == 0) Cleanup();
	
    // Initialize input vectors
    RandomInit(h_A, N);
    RandomInit(h_B, N);	
	
    // Allocate vectors in device memory
    CUDA_SAFE_CALL( hipMalloc((void**)&d_A, size) );
    CUDA_SAFE_CALL( hipMalloc((void**)&d_B, size) );
    CUDA_SAFE_CALL( hipMalloc((void**)&d_C, size) );	

	// DONE: Allocate device memory for input array 'd' and output array 'e'
	CUDA_SAFE_CALL(hipMalloc((void**)&d_D, size));
	CUDA_SAFE_CALL(hipMalloc((void**)&d_E, size));

    // Copy vectors from host memory to device memory
    CUDA_SAFE_CALL( hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice) );
    CUDA_SAFE_CALL( hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice) );

	// DONE: Fill d[] with random data and copy it to the device
	RandomInit(h_D, N);
	CUDA_SAFE_CALL(hipMemcpy(d_D, h_D, size, hipMemcpyHostToDevice));
	
	// Invoke kernel
	//VecAdd<<<1, N>>>(d_A, d_B, d_C);
	VecAdd2<<<1, N>>>(d_A, d_B, d_C, d_D, d_E);
	
#ifdef _DEBUG
	CUDA_SAFE_CALL( hipDeviceSynchronize() );
#endif


	// Copy result from device memory to host memory
    // h_C contains the result in host memory
    CUDA_SAFE_CALL( hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost) );

	// DONE: Copy result E from device to the host
	CUDA_SAFE_CALL(hipMemcpy(h_E, d_E, size, hipMemcpyDeviceToHost));

    // Verify result
	// DONE: Print out E and verify the result.
    int i = 0;
    for (i = 0; i < N; ++i) 
	{
        float sum = h_A[i] + h_B[i];
		float sum2 = h_A[i] + h_B[i] * h_D[i];
		printf("%f + %f = %f\n", h_A[i], h_B[i], h_C[i]);
		printf("%f + %f * %f = %f\n", h_A[i], h_B[i], h_D[i], h_E[i]);
        if (fabs(h_C[i] - sum) > 1e-5)
            break;
		if (fabs(h_E[i] - sum2) > 1e-5)
			break;
    }
    printf("%s \n", (i == N) ? "PASSED" : "FAILED");
    
    Cleanup();
}

void Cleanup(void)
{
    // Free device memory
    if (d_A)
        hipFree(d_A);
    if (d_B)
        hipFree(d_B);
    if (d_C)
        hipFree(d_C);

	// DONE: Free device memory of D and E.
	if (d_D)
		hipFree(d_D);
	if (d_E)
		hipFree(d_E);

    // Free host memory
    if (h_A)
        free(h_A);
    if (h_B)
        free(h_B);
    if (h_C)
        free(h_C);

	// DONE: Free host memory of D and E.	
	if (h_D)
		free(h_D);
	if (h_E)
		free(h_E);
        
    CUDA_SAFE_CALL( hipDeviceReset() );
        
    printf("\nPress ENTER to exit...\n");
    fflush( stdout);
    fflush( stderr);
    getchar();    

    exit(0);
}

// Allocates an array with random float entries.
void RandomInit(float* data, int n)
{
    for (int i = 0; i < n; ++i)
        data[i] = rand() / (float)RAND_MAX;
}
