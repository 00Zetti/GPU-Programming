#include "hip/hip_runtime.h"

#include "common.h"
#include <stdlib.h>
#include <GL/freeglut.h>
#include <math.h>


#define DIM 512
#define blockSize 8
#define blurRadius 6
#define effectiveBlockSize (blockSize+2*blurRadius)

float sourceColors[DIM*DIM];
float *sourceDevPtr;
float *targetDevPtr;

float readBackPixels[DIM*DIM];

// DONE: time addicted variable
int a = 0;

// DONE: declare new texture memory
//texture<float> tex;

void keyboard(unsigned char key, int x, int y)
{
	switch (key)
	{
	case 27:
		exit(0);
		break;
	}
	glutPostRedisplay();
}

// Kernel
// DONE: implement a transformation kernel (diagonal shift/translation)
__global__ void transform(float* sourceDevPtr, float* targetDevPtr, int a)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int index2 = index;

	// translate in diagonal direction (use pixel as vector)
	int2 pixelPos = { (threadIdx.x + a) % DIM, (blockIdx.x + a) % DIM };

	// boarders
	if (pixelPos.x > DIM && pixelPos.x < 0)
		pixelPos.x = 0 ;
	
	if (pixelPos.y > DIM && pixelPos.y < 0)
		pixelPos.y = 0;
		
	// interesting fact: the performance dumps if you proof the negation:
	// if (!(termn)) ...

	// convert into 1d
	index2 = pixelPos.x + pixelPos.y * blockDim.x;

	targetDevPtr[index] = sourceDevPtr[index2];

}

void display(void)	
{
	glClearColor(0.0f, 0.0f, 0.0f, 1.0f);
	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

	// DONE: Transformationskernel auf sourceDevPtr anwenden
	transform <<< DIM, DIM >>>(sourceDevPtr, targetDevPtr, a);
	a++;

	// TODO: Zeitmessung starten (see hipEventCreate, hipEventRecord)

	// TODO: Kernel mit Blur-Filter ausf�hren.

	// TODO: Zeitmessung stoppen und fps ausgeben (see hipEventSynchronize, hipEventElapsedTime, hipEventDestroy)

	// Ergebnis zur CPU zuruecklesen
    CUDA_SAFE_CALL( hipMemcpy( readBackPixels,
                              targetDevPtr,
                              DIM*DIM*4,
                              hipMemcpyDeviceToHost ) );

	// Ergebnis zeichnen (ja, jetzt gehts direkt wieder zur GPU zurueck...) 
	glDrawPixels( DIM, DIM, GL_LUMINANCE, GL_FLOAT, readBackPixels );
	glutSwapBuffers();
}

// clean up memory allocated on the GPU
void cleanup() {
    CUDA_SAFE_CALL( hipFree( sourceDevPtr ) );     

	// TODO: Aufr�umen zus�tzlich angelegter Ressourcen.
//	CUDA_SAFE_CALL(hipUnbindTexture(tex));
	CUDA_SAFE_CALL(hipFree(targetDevPtr));
	CUDA_SAFE_CALL(hipFree(readBackPixels));


}

int main(int argc, char **argv)
{
	glutInit(&argc, argv);
	glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGB | GLUT_DEPTH);
	glutInitWindowSize(DIM, DIM);
	glutCreateWindow("Memory Types");
	glutKeyboardFunc(keyboard);
	glutIdleFunc(display);
	glutDisplayFunc(display);

	// mit Schachbrettmuster f�llen
	for (int i = 0 ; i < DIM*DIM ; i++) {

		int x = (i % DIM) / (DIM/8);
		int y = (i / DIM) / (DIM/8);

		if ((x + y) % 2)
			sourceColors[i] = 1.0f;
		else
			sourceColors[i] = 0.0f;
	}

	// alloc memory on the GPU
	CUDA_SAFE_CALL( hipMalloc( (void**)&sourceDevPtr, DIM*DIM*4 ) );
    CUDA_SAFE_CALL( hipMemcpy( sourceDevPtr, sourceColors, DIM*DIM*4, hipMemcpyHostToDevice ) );

	// DONE: Weiteren Speicher auf der GPU f�r das Bild nach der Transformation und nach dem Blur allokieren.
	// hipMalloc( (void**)&devPtr, imageSize );
	CUDA_SAFE_CALL(hipMalloc((void**)&targetDevPtr, DIM*DIM * 4));
	CUDA_SAFE_CALL(hipMalloc( (void**)&readBackPixels, DIM*DIM*4) );

	// DONE: Binding des Speichers des Bildes an eine Textur mittels hipBindTexture.
	// hipBindTexture( NULL, texName, devPtr, imageSize );
	//CUDA_SAFE_CALL(hipBindTexture(NULL, tex, sourceDevPtr, sizeof(sourceDevPtr)));


	glutMainLoop();

	cleanup();
}
