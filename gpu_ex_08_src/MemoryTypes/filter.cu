#include "hip/hip_runtime.h"

#include "common.h"
#include <stdlib.h>
#include <GL/freeglut.h>
#include <math.h>


#define DIM 512
#define blockSize 8
#define blurRadius 6
#define effectiveBlockSize (blockSize+2*blurRadius)

float sourceColors[DIM*DIM];
float *sourceDevPtr;
float *targetDevPtr;
float *targetBlurDevPtr;

float readBackPixels[DIM*DIM];

// DONE: time addicted variable
int a = 0;

int kernelsize = 10;

// DONE: declare new texture memory
texture<float> tex;

void keyboard(unsigned char key, int x, int y)
{
	switch (key)
	{
	case 27:
		exit(0);
		break;
	case 43:
		kernelsize++;
		break;
	case 45:
		kernelsize--;
		break;
	}
	glutPostRedisplay();
}

// Kernels
// DONE: implement a transformation kernel (diagonal shift/translation)
__global__ void transform(float* sourceDevPtr, float* targetDevPtr, int a)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int index2 = index;

	// translate in diagonal direction (use pixel as vector)
	int2 pixelPos = { (threadIdx.x + a) % DIM, (blockIdx.x + a) % DIM };

	// borders
	if (pixelPos.x > DIM && pixelPos.x < 0)
		pixelPos.x = 0 ;
	
	if (pixelPos.y > DIM && pixelPos.y < 0)
		pixelPos.y = 0;
		
	// interesting fact: the performance dumps if you proof the negation:
	// if (!(term)) ...

	// convert into 1d
	index2 = pixelPos.x + pixelPos.y * blockDim.x;

	targetDevPtr[index] = sourceDevPtr[index2];

}

// DONE: implement a boxcar filter kernel
__global__ void boxcar(float* targetDevPtr, float* targetBlurDevPtr, int kernelsize)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int index2 = index;

	// use pixel as vector
	int2 pixelPos = { threadIdx.x, blockIdx.x };

	// blurred grey value
	float grey = 0.f;

	if (kernelsize < 2)
		targetBlurDevPtr[index] = targetDevPtr[index];
	else
	{
		// borders	
		for (int i = -(kernelsize + 1) / 2; i <(kernelsize + 1) / 2; i++)	// iterate through kernel columns
		{
			for (int j = -(kernelsize + 1) / 2; j <(kernelsize + 1 )/ 2; j++)	// iterate through kernel rows
			{
				if (pixelPos.x + i <= DIM && pixelPos.x - i >= 0
					&& pixelPos.y + j <= DIM && pixelPos.y - j >= 0)	// zero padding
				{
					// convert into 1d
					index2 = pixelPos.x + i + (pixelPos.y + j) * blockDim.x;

					// add partial grey value to the target value
					grey += (targetDevPtr[index2] / float(kernelsize*kernelsize));

				}
			}
		}
		targetBlurDevPtr[index] = grey;
	}
}



// TODO: implement a boxcar filter kernel using texture memory
__global__ void boxcarTex(float* targetBlurDevPtr, int kernelsize)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int index2 = index;

	// use pixel as vector
	int2 pixelPos = { threadIdx.x, blockIdx.x };

	// blurred grey value
	float grey = 0.f;

	if (kernelsize < 2)
		targetBlurDevPtr[index] = tex1Dfetch(tex, index2);
	else
	{
		// borders	
		for (int i = -(kernelsize + 1) / 2; i <(kernelsize + 1) / 2; i++)	// iterate through kernel columns
		{
			for (int j = -(kernelsize + 1) / 2; j <(kernelsize + 1) / 2; j++)	// iterate through kernel rows
			{
				if (pixelPos.x + i <= DIM && pixelPos.x - i >= 0
					&& pixelPos.y + j <= DIM && pixelPos.y - j >= 0)	// zero padding
				{
					// convert into 1d
					index2 = pixelPos.x + i + (pixelPos.y + j) * blockDim.x;

					// add partial grey value to the target value
					grey += (tex1Dfetch(tex, index2) / float(kernelsize*kernelsize));

				}
			}
		}
		targetBlurDevPtr[index] = grey;
	}
}

__global__ void boxcarShared(float* targetDevPtr, float* targetBlurDevPtr, int kernelsize)
{
	//declare variable for shared memory
	
	__shared__ float cache[DIM];

	int tidX = threadIdx.x + blockIdx.x * blockDim.x;
	int tidY = threadIdx.y + blockIdx.y * blockDim.y;
	int tid = tidX + tidY * blockDim.x * gridDim.x;

	int cacheIndex = threadIdx.x;

	//copy data to shared memory
	cache[cacheIndex] = targetDevPtr[tid];

	/*if (threadIdx.x > kernelsize / 2 && threadIdx.x < size - kernelsize / 2 && threadIdx.y > kernelsize / 2 && threadIdx.y < size - kernelsize / 2)
	{
	
	// borders	
		for (int i = -(kernelsize + 1) / 2; i < (kernelsize + 1) / 2; i++)	// iterate through kernel columns
		{
			for (int j = -(kernelsize + 1) / 2; j < (kernelsize + 1) / 2; j++)	// iterate through kernel rows
			{
				if (pixelPos.x + i <= DIM && pixelPos.x - i >= 0
					&& pixelPos.y + j <= DIM && pixelPos.y - j >= 0)	// zero padding
				{
					// convert into 1d
					tid2 = pixelPos.x + i + (pixelPos.y + j);

					// add partial grey value to the target value
					grey += (cache[tid2] / float(kernelsize*kernelsize));

				}
			}
		}

	}*/

	targetBlurDevPtr[tid] = cache[cacheIndex];		
}

void display(void)	
{
	glClearColor(0.0f, 0.0f, 0.0f, 1.0f);
	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

	// DONE: Transformationskernel auf sourceDevPtr anwenden
	transform <<< DIM, DIM >>>(sourceDevPtr, targetDevPtr, a);
	a++;

	// DONE: Zeitmessung starten (see hipEventCreate, hipEventRecord)
	hipEvent_t start, stop;
	float time;
	CUDA_SAFE_CALL(hipEventCreate(&start));
	CUDA_SAFE_CALL(hipEventCreate(&stop));
	CUDA_SAFE_CALL(hipEventRecord(start, 0));

	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);
	float maxThreadsPerBlock = prop.maxThreadsPerBlock;

	// DONE: Kernel mit Blur-Filter ausf�hren.
	//boxcar <<< DIM, DIM >>>(targetDevPtr, targetBlurDevPtr, kernelsize);
	//boxcarTex <<< DIM, DIM >> >(targetBlurDevPtr, kernelsize);
	boxcarShared <<<DIM,DIM>>>(targetDevPtr, targetBlurDevPtr,kernelsize);



	// DONE: Zeitmessung stoppen und fps ausgeben (see hipEventSynchronize, hipEventElapsedTime, hipEventDestroy)
	CUDA_SAFE_CALL(hipEventRecord(stop, 0));
	CUDA_SAFE_CALL(hipEventSynchronize(stop));
	CUDA_SAFE_CALL(hipEventElapsedTime(&time, start, stop));
	CUDA_SAFE_CALL(hipEventDestroy(start));
	CUDA_SAFE_CALL(hipEventDestroy(stop));
	printf("Elapsed time: %f ms\n", time);
	printf("Kernelsize: %u\n\n", kernelsize);

	// Ergebnis zur CPU zuruecklesen
    //CUDA_SAFE_CALL( hipMemcpy( readBackPixels, targetDevPtr, DIM*DIM*4, hipMemcpyDeviceToHost ) ); // task01	
	CUDA_SAFE_CALL(hipMemcpy(readBackPixels, targetBlurDevPtr, DIM*DIM * 4, hipMemcpyDeviceToHost));

	// Ergebnis zeichnen (ja, jetzt gehts direkt wieder zur GPU zurueck...) 
	glDrawPixels( DIM, DIM, GL_LUMINANCE, GL_FLOAT, readBackPixels );
	glutSwapBuffers();
}
// clean up memory allocated on the GPU
void cleanup() {
    CUDA_SAFE_CALL( hipFree( sourceDevPtr ) );     

	// TODO: Aufr�umen zus�tzlich angelegter Ressourcen.
	CUDA_SAFE_CALL(hipUnbindTexture(tex));
	CUDA_SAFE_CALL(hipFree(targetDevPtr));
	CUDA_SAFE_CALL(hipFree(targetBlurDevPtr));
	CUDA_SAFE_CALL(hipFree(readBackPixels));
}

int main(int argc, char **argv)
{
	glutInit(&argc, argv);
	glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGB | GLUT_DEPTH);
	glutInitWindowSize(DIM, DIM);
	glutCreateWindow("Memory Types");
	glutKeyboardFunc(keyboard);
	glutIdleFunc(display);
	glutDisplayFunc(display);

	// mit Schachbrettmuster f�llen
	for (int i = 0 ; i < DIM*DIM ; i++) {

		int x = (i % DIM) / (DIM/8);
		int y = (i / DIM) / (DIM/8);

		if ((x + y) % 2)
			sourceColors[i] = 1.0f;
		else
			sourceColors[i] = 0.0f;
	}

	// alloc memory on the GPU
	CUDA_SAFE_CALL( hipMalloc( (void**)&sourceDevPtr, DIM*DIM*4 ) );
    CUDA_SAFE_CALL( hipMemcpy( sourceDevPtr, sourceColors, DIM*DIM*4, hipMemcpyHostToDevice ) );

	// DONE: Weiteren Speicher auf der GPU f�r das Bild nach der Transformation und nach dem Blur allokieren.
	// hipMalloc( (void**)&devPtr, imageSize );
	CUDA_SAFE_CALL(hipMalloc((void**)&targetDevPtr, DIM*DIM*4));
	CUDA_SAFE_CALL(hipMalloc((void**)&readBackPixels, DIM*DIM*4));
	CUDA_SAFE_CALL(hipMalloc((void**)&targetBlurDevPtr, DIM*DIM*4));

	// DONE: Binding des Speichers des Bildes an eine Textur mittels hipBindTexture.
	//hipBindTexture( NULL, texName, devPtr, imageSize ); // use direct size, not sizeof()!!!
	CUDA_SAFE_CALL(hipBindTexture(NULL, tex, targetDevPtr, DIM*DIM*4));

	
	glutKeyboardFunc(keyboard);
	glutMainLoop();

	cleanup();
}
